#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "file.hpp"
#include "holders.hpp"

int main(int __argc,char *__argv[]){
    if(__argc != 2){
        printf("there must be 1 argument");
        return -1;
    }
    neuralnetwork NN = FromFile(__argv[0]);
    int NumofFirst = NN.layers[0].NumOfNu;
    int NumofLast = NN.layers[NN.NumOfLayers-1].NumOfNu;
    neuralnetwork* Nptr;
    hipMallocManaged((void**)&Nptr,sizeof(neuralnetwork));
    Nptr = &NN;
    

    hipFree(Nptr);
    return 0;
}