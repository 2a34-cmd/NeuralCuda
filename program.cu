#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "file.hpp"
#include "holders.hpp"
#include "mnist.hpp"
#include "calc.cu"

int main(int __argc,char *__argv[]){
    if(__argc != 5){
        printf("there must be 5 arguments");
        return -1;
    }
    neuralnetwork NN = FromFile(__argv[0]);
    int NumofFirst = NN.layers[0].NumOfNu;
    int NumofLast = NN.layers[NN.NumOfLayers-1].NumOfNu;
    neuralnetwork* Nptr;
    hipMallocManaged((void**)&Nptr,sizeof(neuralnetwork));
    Nptr = &NN;
    byte** ITNN = InputsToNN(__argv[1],(int)__argv[3]);
    byte** EFNN = ExpectedFromNN(__argv[2],(int)__argv[3]);
    for (size_t i = (int)__argv[3]; i < sizeof(ITNN)/sizeof(*ITNN); i++)
    {
        InputFirst<<<>>>(Nptr,ITNN[i]);
        hipDeviceSynchronize();
        for (size_t j = 1; j < Nptr->NumOfLayers; j++)
        {
            calc<<<>>>(Nptr,j);
            hipDeviceSynchronize();
        }
        diffLast(Nptr,EFNN[i],(double)__argv[4]);
        hipDeviceSynchronize();
        for (size_t j = Nptr->NumOfLayers; j >= 1; j--)
        {
            diffcalc<<<>>>(Nptr,j);
            hipDeviceSynchronize();
        }
        back(Nptr);
    }
    

    hipFree(Nptr);
    return 0;
}