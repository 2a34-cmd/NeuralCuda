#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>




#define N 10000
#define S N*sizeof(int)


// kerenl, the parallelized proccess
__global__ void VAdding(int *a, int *b, int *c){
    int i = threadIdx.x;
    int j = blockIdx.x;
    int globalindex = blockDim.x * j + i;
    c[globalindex] = a[globalindex] + b[globalindex];

}

int main(void)
{
   
   int a[N];
   int b[N];
   int c[N];

   int *Ca,*Cb,*Cc;
   for(int i = 0; i <N;i++){
     a[i]= -i;
     b[i]= i*i;
   } 

   hipMalloc((void**)&Ca,S); 
   hipMalloc((void**)&Cb,S); 
   hipMalloc((void**)&Cc,S); 


   hipMemcpy(Ca,a, S,hipMemcpyHostToDevice);
   hipMemcpy(Cb,b, S,hipMemcpyHostToDevice); 

    VAdding<<<100,100>>>(Ca,Cb,Cc);

    hipMemcpy(c,Cc, S,hipMemcpyDeviceToHost);


    for(int i = 0; i<N; i+=100){
        
        printf("%d + %d = %d \n",a[i],b[i],c[i]);
    }

    
    hipFree(Ca);
    hipFree(Cb);
    hipFree(Cc);

    return 0;
}


