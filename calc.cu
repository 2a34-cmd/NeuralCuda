#include "hip/hip_runtime.h"
#include "file.cpp"
#include <math.h>

__global__ void calc(struct neuralnetwork* neuralnetptr,int layerIndex,double* inputs,double* outputs){
    int j = blockDim.x * blockIdx.x + threadIdx.x;//index of neuron

    neuralnetwork NN = *neuralnetptr;
    connection* conptr;
    NuCon Froms = NN.layers[layerIndex].group[j].froms;
    hipMalloc((void**)&conptr,Froms.NumOfCon*sizeof(connection));

    double weitghedSum = 0;
    for (int k = 0; k < Froms.NumOfCon; k++)
    {

        weitghedSum+=  *  inputs[k];
    }
    weitghedSum += NN.layers[layerIndex].group[j].bias;
    Activation(&weitghedSum,NN.ActivFunc,&(outputs[j]));
    NN.layers[layerIndex].group[j].value = outputs[j];
    hipFree(conptr);
}
__global__ void diffcalc(neuralnetwork* neuralnetptr,int layerIndex,double* Expected,double* outputs){
    int i = blockDim.x * blockIdx.x + threadIdx.x;//index of neuron


}
__device__ void Activation(double* input,ActivationFunc af,double* output){
    switch (af)
    {
    case 1:
        double out = tanh(*input);
        output = &out;
        break;
    case 2:
        double out = exp(*input)/(1+exp(*input));
        output = &out;
        break;
    case 3:
        double out=0;
        if(*input > 0) out = *input;
        output = &out;
        break;
    case 4:
        output = input;
        break;
    default:
        break;
    }
}
__device__ void DActivation(double* input,ActivationFunc AF,double* output){
    switch(AF){
    case 1:
        double out = pow(cosh(*input),-2);
        output = &out;
        break;
    case 2:
        double out = exp(*input)*pow(1+exp(*input),-2);
        output = &out;
        break;
    case 3:
        double out=0;
        if(*input > 0) out = 1;
        output = &out;
        break;
    case 4:
        double out = 1;
        output = &out;
        break;
    default:
        break;
    }
}