#include "hip/hip_runtime.h"
#include <math.h>
#include "holders.hpp"

/// The functions below don't use the idea of child threads, which may be implemented later

// this function needs <<<N,M>>> where N*M equals number of neuronals in the layer with layerIndex  
// can't use with layerIndex == 0
/// to call it, you must call InputFirst
__global__ void calc(neuralnetwork* neuralnetptr,int layerIndex){
    int j = blockDim.x * blockIdx.x + threadIdx.x;//index of neuron
    neuralnetwork NN = *neuralnetptr;
    NuCon Froms = NN.layers[layerIndex].group[j].froms;
    double weightedSum = NN.layers[layerIndex].group[j].bias;
    for (size_t i = 0; i < Froms.NumOfCon; i++)
    {
        weightedSum += Froms.ConPtr[i]->weight * NN.layers[Froms.ConPtr[i]->LF].group[Froms.ConPtr[i]->FromId].value;
    }
    Activation(weightedSum,NN.ActivFunc,&NN.layers[layerIndex].group[j].value);
    
}
// this function needs <<<N,M>>> where N*M equals number of neuronals in the layer with layerIndex
// can't use the function with layerIndex +1 == neuralnetptr->numOfLayers
/// to call it, you must call difflast and calc
__global__ void diffcalc(neuralnetwork* neuralnetptr,int layerIndex){
    int j = blockDim.x * blockIdx.x + threadIdx.x;//index of neuron

    neuralnetwork NN = *neuralnetptr;
    connection* conptr;
    NuCon Toes = NN.layers[layerIndex].group[j].toes;
    double value = NN.layers[layerIndex].group[j].value;
    double weightedSum = 0;
    double Term = 1;
    for (size_t i = 0; i < Toes.NumOfCon; i++)
    {
        Term *= DActivation(Toes.ConPtr[i]->weight * value + NN.layers[Toes.ConPtr[i]->LT].group[Toes.ConPtr[i]->ToId].bias,
                            NN.ActivFunc);
        Term *= NN.layers[Toes.ConPtr[i]->LT].group[Toes.ConPtr[i]->ToId].difference;
        Term *= Toes.ConPtr[i]->weight;
        weightedSum += Term;
        Term = 1;
    }
    NN.layers[layerIndex].group[j].difference = weightedSum;
}


__global__ void InputFirst(neuralnetwork* neuralnetptr, double* Inputs){
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    neuralnetptr->layers[0].group[j].value = Inputs[j];
}
__global__ void diffLast(neuralnetwork* neuralnetptr,double* Expected, double MLRate){
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    neuron n = neuralnetptr->layers[neuralnetptr->NumOfLayers-1].group[j];
    n.difference = MLRate * abs(n.value - Expected[j]);
}

//<<<N,M>>> where N*M == number of neurons and connections
/// to call it, you must call cycle and its formers
__global__ void back(neuralnetwork* neuralnetptr){
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    neuralnetwork NN = *neuralnetptr;
    if(j<NN.NumOfConnenction){
        neuron n = NN.layers[NN.connections->LT].group[NN.connections->ToId];
        double diff = n.difference,val = n.value;
        NN.connections[j].weight -= diff*val;
    }
    else{   
        j -= NN.NumOfConnenction;
        int k =j;
        int x = j,y=0;
        for (size_t i = 0; i < NN.NumOfLayers; i++)
        {
            if(NN.layers[i].NumOfNu < k){
                k-= NN.layers[i].NumOfNu;
                x=k;
                y++;
            }else{
                x--;
                break;
            }
        }
        NN.layers[y].group[x].bias -= NN.layers[y].group[x].difference;
    }
}


//the activation function and its derviative
__device__ void Activation(double input,ActivationFunc af,double* output){
    switch (af)
    {
    case 1:
        double out = tanh(input);
        output = &out;
        break;
    case 2:
        double out = exp(input)/(1+exp(input));
        output = &out;
        break;
    case 3:
        double out=0;
        if(input > 0) out = input;
        output = &out;
        break;
    case 4:
        output = &input;
        break;
    default:
        break;
    }
}
__device__ double DActivation(double input,ActivationFunc AF){
    switch(AF){
    case 1:
        return pow(cosh(input),-2);
    case 2:
        return exp(input)*pow(1+exp(input),-2);
    case 3:
        if(input > 0) return 1;
        else return 0;
    case 4:
        return 1;
    default:
        return NAN;
    }
}