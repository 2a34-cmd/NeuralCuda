#include "hip/hip_runtime.h"
#include "file.cpp"
#include <math.h>

__global__ void calc(struct neuralnetwork* neuralnetptr,int layerIndex,double* inputs,double* outputs){
    int i = threadIdx.x;//index of connection
    int j = blockIdx.x;//index of nueron

    neuralnetwork NN = *neuralnetptr;
    connection* conptr;
    int* Froms;
    FromsOfNeuron<<<2,112901>>>(conptr,layerIndex,j,Froms);
    hipMalloc((void**)&conptr,(*Froms)*sizeof(connection));

    double weitghedSum = 0;
    for (int k = 0; k < *Froms; k++)
    {
        weitghedSum+= conptr[i].weight * inputs[i];
    }
    double* param = outputs + (sizeof(double) * j);
    Activation(&weitghedSum,NN.ActivFunc,param);
    NN.layers[layerIndex].group[j].value = *param;
    hipFree(Froms);
}
__global__ void train(neuralnetwork* neuralnetptr,double* inputs,double* outputs){
    int i = threadIdx.x;
    int j = blockIdx.x;
    int globalindex = blockDim.x * j + i;


}

__device__ void FromsOfNeuron(connection* conptr,unsigned int LId,unsigned int NId,int* output){
    int i = threadIdx.x;
    int j = blockIdx.x;
    int globalindex = blockDim.x * j + i;
    if(conptr[globalindex].LT == LId && conptr[globalindex].ToId == NId){
        (*output)++;
    }
}
__device__ void ToesOfNeuron(connection* conptr,unsigned int LId,unsigned int NId,int* output){
    int i = threadIdx.x;
    int j = blockIdx.x;
    int globalindex = blockDim.x * j + i;
    if(conptr[globalindex].LF == LId && conptr[globalindex].FromId == NId){
        (*output)++;
    }
}
__device__ void Activation(double* input,ActivationFunc af,double* output){
    switch (af)
    {
    case 1:
        double out = tanh(*input);
        output = &out;
        break;
    case 2:
        double out = exp(*input)/(1+exp(*input));
        output = &out;
        break;
    case 3:
        double out=0;
        if(*input > 0) out = *input;
        output = &out;
        break;
    case 4:
        output = input;
        break;
    default:
        break;
    }
}